#include "hip/hip_runtime.h"
/*******************************************************************************/
/*  StePS - STEreographically Projected cosmological Simulations                */
/*    Copyright (C) 2017-2022 Gabor Racz                                        */
/*                                                                              */
/*    This program is free software; you can redistribute it and/or modify      */
/*    it under the terms of the GNU General Public License as published by      */
/*    the Free Software Foundation; either version 2 of the License, or         */
/*    (at your option) any later version.                                       */
/*                                                                              */
/*    This program is distributed in the hope that it will be useful,           */
/*    but WITHOUT ANY WARRANTY; without even the implied warranty of            */
/*    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the             */
/*    GNU General Public License for more details.                              */
/********************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <omp.h>
#include <time.h>
#include "mpi.h"
#include "global_variables.h"
#include "hip/hip_runtime.h"
#include ""

#define BLOCKSIZE 256


extern int e[2202][4];
extern REAL w[3];
extern int N, el;

#ifndef PERIODIC
hipError_t forces_cuda(REAL*x, REAL*F, int n_GPU, int ID_min, int ID_max);
#else
int ewald_space(REAL R, int ewald_index[2102][4]);
hipError_t forces_periodic_cuda(REAL*x, REAL*F, int n_GPU, int ID_min, int ID_max);
#endif

#ifndef PERIODIC
void forces(REAL*x, REAL*F, int ID_min, int ID_max)
{
	forces_cuda(x, F, n_GPU, ID_min, ID_max);
	return;
}
#endif
#ifdef PERIODIC
void forces_periodic(REAL*x, REAL*F, int ID_min, int ID_max)
{
	forces_periodic_cuda(x, F, n_GPU, ID_min, ID_max);
	return;
}
#endif


void recalculate_softening();

#ifndef PERIODIC
__global__ void ForceKernel(int n, const int N, const REAL *xx, const REAL *xy, const REAL *xz, REAL *F, const REAL* M, const REAL* SOFT_LENGTH, const REAL mass_in_unit_sphere, const REAL DE, const int COSMOLOGY, const int COMOVING_INTEGRATION, int ID_min, int ID_max)
{
	REAL Fx_tmp, Fy_tmp, Fz_tmp;
	REAL r, dx, dy, dz, wij, beta_priv, beta_privp2;
	REAL SOFT_CONST[5];
	int i, j, id;
	id = blockIdx.x * blockDim.x + threadIdx.x;
	Fx_tmp = Fy_tmp = Fz_tmp = 0.0;
	for (i = (ID_min+id); i<=ID_max; i+=n)
		{
			for (j = 0; j<N; j++)
			{
				beta_priv = (SOFT_LENGTH[i]+SOFT_LENGTH[j]);
				beta_privp2 = beta_priv*0.5;
				//calculating particle distances
				dx = (xx[j] - xx[i]);
				dy = (xy[j] - xy[i]);
				dz = (xz[j] - xz[i]);
				r = sqrt(pow(dx, 2) + pow(dy, 2) + pow(dz, 2));
				if (r >= beta_priv)
				{
					wij = M[j] / (pow(r, 3));
				}
				else if (r > beta_privp2 && r < beta_priv)
				{
					SOFT_CONST[0] = -32.0/(3.0*pow(beta_priv, 6));
					SOFT_CONST[1] = 38.4/pow(beta_priv, 5);
					SOFT_CONST[2] = -48.0/pow(beta_priv, 4);
					SOFT_CONST[3] = 64.0/(3.0*pow(beta_priv, 3));
					SOFT_CONST[4] = -1.0/15.0;
					wij = M[j]*(SOFT_CONST[0] * pow(r, 3) + SOFT_CONST[1] * pow(r, 2) + SOFT_CONST[2] * r + SOFT_CONST[3] + SOFT_CONST[4] / pow(r, 3));
				}
				else
				{
					SOFT_CONST[0] = 32.0/pow(beta_priv, 6);
					SOFT_CONST[1] = -38.4/pow(beta_priv, 5);
					SOFT_CONST[2] = 32.0/(3.0*pow(beta_priv, 3));
					wij = M[j]*(SOFT_CONST[0] * pow(r, 3) + SOFT_CONST[1] * pow(r, 2) + SOFT_CONST[2]);
				}
				Fx_tmp += wij*(dx);
				Fy_tmp += wij*(dy);
				Fz_tmp += wij*(dz);

			}
			if(COSMOLOGY == 1 && COMOVING_INTEGRATION == 1)//Adding the external force from the outside of the simulation volume, if we run non-periodic comoving cosmological simulation
			{
				Fx_tmp += mass_in_unit_sphere * xx[i];
				Fy_tmp += mass_in_unit_sphere * xy[i];
				Fz_tmp += mass_in_unit_sphere * xz[i];
			}
			else if(COSMOLOGY == 1 && COMOVING_INTEGRATION == 0)
			{
				Fx_tmp += DE * xx[i];
				Fy_tmp += DE * xy[i];
				Fz_tmp += DE * xz[i];
			}
			F[3*(i-ID_min)] += Fx_tmp;
			F[3*(i-ID_min)+1] += Fy_tmp;
			F[3*(i-ID_min)+2] += Fz_tmp;
			Fx_tmp = Fy_tmp = Fz_tmp = 0.0;

		}
}
#endif

#ifdef PERIODIC
__global__ void ForceKernel_periodic(int n, int N, const REAL *xx, const REAL *xy, const REAL *xz, REAL *F, const int IS_PERIODIC, const REAL* M, const REAL* SOFT_LENGTH, const REAL L, const int *e, int el, int ID_min, int ID_max)
{
	REAL Fx_tmp, Fy_tmp, Fz_tmp;
	REAL r, dx, dy, dz, wij, beta_priv, beta_privp2;
	REAL SOFT_CONST[5];
	int i, j, m, id;

	id = blockIdx.x * blockDim.x + threadIdx.x;
	Fx_tmp = Fy_tmp = Fz_tmp = 0;
	if (IS_PERIODIC == 1)
	{
		for (i = (ID_min+id); i<=ID_max; i+=n)
		{
			for (j = 0; j<N; j++)
			{
				beta_priv = (SOFT_LENGTH[i]+SOFT_LENGTH[j]);
				beta_privp2 = beta_priv*0.5;
				//calculating particle distances
				dx = (xx[j] - xx[i]);
				dy = (xy[j] - xy[i]);
				dz = (xz[j] - xz[i]);
				//in this quasi-periodic caes, we use only the nearest image
				if (fabs(dx)>0.5*L)
					dx = dx - L*dx / fabs(dx);
				if (fabs(dy)>0.5*L)
					dy = dy - L*dy / fabs(dy);
				if (fabs(dz)>0.5*L)
					dz = dz - L*dz / fabs(dz);
				r = sqrt(pow(dx, 2) + pow(dy, 2) + pow(dz, 2));
                                wij = 0.0;
				if (r >= beta_priv)
				{
					wij = M[j] / (pow(r, 3));
				}
				else if (r > beta_privp2 && r < beta_priv)
                                {
					SOFT_CONST[0] = -32.0/(3.0*pow(beta_priv, 6));
					SOFT_CONST[1] = 38.4/pow(beta_priv, 5);
					SOFT_CONST[2] = -48.0/pow(beta_priv, 4);
					SOFT_CONST[3] = 64.0/(3.0*pow(beta_priv, 3));
					SOFT_CONST[4] = -1.0/15.0;
					wij = M[j]*(SOFT_CONST[0] * pow(r, 3) + SOFT_CONST[1] * pow(r, 2) + SOFT_CONST[2] * r + SOFT_CONST[3] + SOFT_CONST[4] / pow(r, 3));
				}
				else
				{
					SOFT_CONST[0] = 32.0/pow(beta_priv, 6);
					SOFT_CONST[1] = -38.4/pow(beta_priv, 5);
					SOFT_CONST[2] = 32.0/(3.0*pow(beta_priv, 3));
					wij = M[j]*(SOFT_CONST[0] * pow(r, 3) + SOFT_CONST[1] * pow(r, 2) + SOFT_CONST[2]);
				}
				Fx_tmp += wij*(dx);
				Fy_tmp += wij*(dy);
				Fz_tmp += wij*(dz);

			}
			F[3*(i-ID_min)] += Fx_tmp;
			F[3*(i-ID_min)+1] += Fy_tmp;
			F[3*(i-ID_min)+2] += Fz_tmp;
			Fx_tmp = Fy_tmp = Fz_tmp = 0.0;
		}
	}
	else if (IS_PERIODIC >= 2)
	{
		for (i = (ID_min+id); i<=ID_max; i=i+n)
		{
			for (j = 0; j<N; j++)
			{
				beta_priv = (SOFT_LENGTH[i]+SOFT_LENGTH[j]);
				beta_privp2 = beta_priv*0.5;
				//calculating particle distances
				dx = (xx[j] - xx[i]);
				dy = (xy[j] - xy[i]);
				dz = (xz[j] - xz[i]);
				//in this function we use multiple images
				for (m = 0; m < 3*el; m = m+3)
				{
					r = sqrt(pow((dx - ((REAL)e[m])*L), 2) + pow((dy - ((REAL)e[m+1])*L), 2) + pow((dz-((REAL)e[m+2])*L), 2));
					wij = 0.0;
					if (r >= beta_priv && r < 2.6*L)
					{
						wij = M[j] / (pow(r, 3));
					}
					else if (r > beta_privp2 && r <= beta_priv)
					{
						SOFT_CONST[0] = -32.0/(3.0*pow(beta_priv, 6));
						SOFT_CONST[1] = 38.4/pow(beta_priv, 5);
						SOFT_CONST[2] = -48.0/pow(beta_priv, 4);
						SOFT_CONST[3] = 64.0/(3.0*pow(beta_priv, 3));
						SOFT_CONST[4] = -1.0/15.0;
						wij = M[j]*(SOFT_CONST[0] * pow(r, 3) + SOFT_CONST[1] * pow(r, 2) + SOFT_CONST[2] * r + SOFT_CONST[3] + SOFT_CONST[4] / pow(r, 3));
					}
					else if (r <= beta_privp2)
					{
						SOFT_CONST[0] = 32.0/pow(beta_priv, 6);
						SOFT_CONST[1] = -38.4/pow(beta_priv, 5);
						SOFT_CONST[2] = 32.0/(3.0*pow(beta_priv, 3));
						wij = M[j]*(SOFT_CONST[0] * pow(r, 3) + SOFT_CONST[1] * pow(r, 2) + SOFT_CONST[2]);
					}
					if (wij != 0)
					{
						Fx_tmp += wij*(dx - ((REAL)e[m])*L);
						Fy_tmp += wij*(dy - ((REAL)e[m + 1])*L);
						Fz_tmp += wij*(dz - ((REAL)e[m + 2])*L);
					}
				}

			}
			F[3 * (i-ID_min)] += Fx_tmp;
			F[3 * (i-ID_min) + 1] += Fy_tmp;
			F[3 * (i-ID_min) + 2] += Fz_tmp;
			Fx_tmp = Fy_tmp = Fz_tmp = 0;
		}
	}

}
#endif

void recalculate_softening()
{
	beta = ParticleRadi;
	if(COSMOLOGY ==1)
	{
		rho_part = M_min/(4.0*pi*pow(beta, 3.0) / 3.0);
	}
}

#ifndef PERIODIC
hipError_t forces_cuda(REAL*x, REAL*F, int n_GPU, int ID_min, int ID_max) //Force calculation on GPU
{
	int i, j;
	int mprocessors;
	int GPU_ID, nthreads;
	int N_GPU, GPU_index_min; //number of particles in this GPU, the first particles index
	hipError_t cudaStatus;
	cudaStatus = hipSuccess;
	double omp_start_time, omp_end_time;
	REAL DE = (REAL) H0*H0*Omega_lambda;
	REAL *xx_tmp, *xy_tmp, *xz_tmp, *F_tmp;
	REAL *dev_xx= 0;
	REAL *dev_xy= 0;
	REAL *dev_xz= 0;
	REAL *dev_M = 0;
	REAL *dev_SOFT_LENGTH = 0; //v0.3.7.1
	REAL *dev_F = 0;

	// Get the number of CUDA devices.
	int numDevices;
	hipGetDeviceCount(&numDevices);
	if(numDevices<n_GPU)
	{
		if(numDevices == 1)
			fprintf(stderr, "Error: MPI rank %i: Cannot allocate %i GPUs, because only one is available\n", rank, n_GPU);
		else
			fprintf(stderr, "Error: MPI rank %i: Cannot allocate %i GPUs, because only %i are available\n", rank, n_GPU, numDevices);
		n_GPU = numDevices;
		printf("Number of GPUs set to %i\n", n_GPU);
	}

	if(!(xx_tmp = (REAL*)malloc(N*sizeof(REAL))))
	{
		fprintf(stderr, "MPI task %i: failed to allocate memory for xx_tmp (for CUDA force canculation).\n", rank);
		exit(-2);
	}
	if(!(xy_tmp = (REAL*)malloc(N*sizeof(REAL))))
	{
		fprintf(stderr, "MPI task %i: failed to allocate memory for xy_tmp (for CUDA force canculation).\n", rank);
		exit(-2);
	}
	if(!(xz_tmp = (REAL*)malloc(N*sizeof(REAL))))
	{
		fprintf(stderr, "MPI task %i: failed to allocate memory for xz_tmp (for CUDA force canculation).\n", rank);
		exit(-2);
	}
	for(i = 0; i < N; i++)
	{
		xx_tmp[i] = x[3*i];
		xy_tmp[i] = x[3*i+1];
		xz_tmp[i] = x[3*i+2];
	}
	//timing
	omp_start_time = omp_get_wtime();
	//timing
	omp_set_dynamic(0);		// Explicitly disable dynamic teams
	omp_set_num_threads(n_GPU);	// Use n_GPU threads
#pragma omp parallel default(shared) private(GPU_ID, F_tmp, i, j, mprocessors, cudaStatus, N_GPU, GPU_index_min, nthreads, dev_xx, dev_xy, dev_xz, dev_M, dev_SOFT_LENGTH, dev_F)
{
		#pragma omp critical
		{
		nthreads = omp_get_num_threads();
		GPU_ID = omp_get_thread_num(); //thread ID = GPU_ID
		}
		if(GPU_ID == 0)
		{
			N_GPU = (ID_max-ID_min+1)/n_GPU+(ID_max-ID_min+1)%n_GPU;
			GPU_index_min = ID_min;
		}
		else
		{
			N_GPU = (ID_max-ID_min+1)/n_GPU;
			GPU_index_min = ID_min + (ID_max-ID_min+1)%n_GPU+N_GPU*GPU_ID;
		}
		if(!(F_tmp = (REAL*)malloc(3 * N_GPU * sizeof(REAL))))
		{
			fprintf(stderr, "MPI task %i: failed to allocate memory for F_tmp (for CUDA force canculation).\n", rank);
			exit(-2);
		}
		for(i=0; i < N_GPU; i++)
		{
			for(j=0; j<3; j++)
			F_tmp[3*i + j] = 0.0f;
		}
		//Checking for the GPU
		#pragma omp critical
		hipDeviceGetAttribute(&mprocessors, hipDeviceAttributeMultiprocessorCount, GPU_ID);
		if(GPU_ID == 0)
		{

			printf("MPI task %i: GPU force calculation.\n Number of GPUs: %i\n Number of OMP threads: %i\n Number of threads per GPU: %i\n", rank, n_GPU, nthreads, 32*mprocessors*BLOCKSIZE);
		}
		#pragma omp critical
		cudaStatus = hipSetDevice(GPU_ID); //selecting the GPU
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		// Allocate GPU buffers for coordinate and mass vectors
		cudaStatus = hipMalloc((void**)&dev_xx, N * sizeof(REAL));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: xx hipMalloc failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&dev_xy, N * sizeof(REAL));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: xy hipMalloc failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&dev_xz, N * sizeof(REAL));
		if (cudaStatus != hipSuccess) {
                	fprintf(stderr, "MPI rank %i: GPU%i: xz hipMalloc failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&dev_M, N * sizeof(REAL));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: M hipMalloc failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		// Allocate GPU buffers for the softening vector
		cudaStatus = hipMalloc((void**)&dev_SOFT_LENGTH, N * sizeof(REAL)); //v0.3.7.1
                if (cudaStatus != hipSuccess) {
                        fprintf(stderr, "MPI rank %i: GPU%i: SOFT_LENGTH hipMalloc failed!\n", rank, GPU_ID);
			ForceError = true;
                        goto Error;
                }
		// Allocate GPU buffers for force vectors
		cudaStatus = hipMalloc((void**)&dev_F, 3 * N_GPU * sizeof(REAL));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: F hipMalloc failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_xx, xx_tmp, N * sizeof(REAL), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: hipMemcpy xx in failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		cudaStatus = hipMemcpy(dev_xy, xy_tmp, N * sizeof(REAL), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: hipMemcpy xy in failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		cudaStatus = hipMemcpy(dev_xz, xz_tmp, N * sizeof(REAL), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: hipMemcpy xz in failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		cudaStatus = hipMemcpy(dev_M, M, N * sizeof(REAL), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: hipMemcpy M in failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		cudaStatus = hipMemcpy(dev_SOFT_LENGTH, SOFT_LENGTH, N * sizeof(REAL), hipMemcpyHostToDevice); // v0.3.7.1
                if (cudaStatus != hipSuccess) {
                        fprintf(stderr, "MPI rank %i: GPU%i: hipMemcpy SOFT_LENGTH in failed!\n", rank, GPU_ID);
			ForceError = true;
                        goto Error;
                }
		cudaStatus = hipMemcpy(dev_F, F_tmp, 3 * N_GPU * sizeof(REAL), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: hipMemcpy F in failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		printf("MPI task %i: GPU%i: ID_min = %i\tID_max = %i\n", rank, GPU_ID, GPU_index_min, GPU_index_min+N_GPU-1);
		// Launch a kernel on the GPU
		ForceKernel<<<32*mprocessors, BLOCKSIZE>>>(32 * mprocessors * BLOCKSIZE, N, dev_xx, dev_xy, dev_xz, dev_F, dev_M, dev_SOFT_LENGTH, mass_in_unit_sphere, DE, COSMOLOGY, COMOVING_INTEGRATION, GPU_index_min, GPU_index_min+N_GPU-1);
		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: ForceKernel launch failed: %s\n", rank, GPU_ID, hipGetErrorString(cudaStatus));
			ForceError = true;
			goto Error;
		}
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: hipDeviceSynchronize returned error code %d after launching ForceKernel!\n", rank, GPU_ID, cudaStatus);
			ForceError = true;
			goto Error;
		}
		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(F_tmp, dev_F, 3 * N_GPU * sizeof(REAL), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI %i: GPU%i: hipMemcpy F out failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		if(GPU_ID == 0)
		{
			for (i = 0; i < N_GPU; i++)
			{
				for (j = 0; j < 3; j++)
				{
					F[3*i+j] = F_tmp[(3 * i) + j];
				}
			}
		}
		else
		{
			for (i = GPU_index_min; i < GPU_index_min + N_GPU; i++)
			{
				for (j = 0; j < 3; j++)
				{
					F[3*(i-ID_min)+j] = F_tmp[3 * (i-GPU_index_min) + j];
				}
			}
		}
	free(F_tmp);
	Error:
		hipFree(dev_xx);
                hipFree(dev_xy);
                hipFree(dev_xz);
                hipFree(dev_M);
                hipFree(dev_F);
		hipFree(dev_SOFT_LENGTH);
		hipDeviceReset();

}
	free(xx_tmp);
	free(xy_tmp);
	free(xz_tmp);
	//timing
	omp_end_time = omp_get_wtime();
	//timing
	printf("Force calculation finished on MPI task %i. Force calculation wall-clock time = %fs.\n", rank, omp_end_time-omp_start_time);
	return cudaStatus;
}
#endif

#ifdef PERIODIC
hipError_t forces_periodic_cuda(REAL*x, REAL*F, int n_GPU, int ID_min, int ID_max) //Force calculation with multiple images on GPU
{
	int i, j;
	int mprocessors;
	int GPU_ID, nthreads;
	int N_GPU, GPU_index_min; //number of particles in this GPU, the first particles index
	hipError_t cudaStatus;
	cudaStatus = hipSuccess;
	double omp_start_time, omp_end_time;
	REAL *xx_tmp, *xy_tmp, *xz_tmp, *F_tmp;
	REAL *dev_xx= 0;
	REAL *dev_xy= 0;
	REAL *dev_xz= 0;
	REAL *dev_M = 0;
	REAL *dev_SOFT_LENGTH = 0;
	REAL *dev_F = 0;
	int *dev_e;
	int e_tmp[6606];

	int numDevices;
	hipGetDeviceCount(&numDevices);
	if(numDevices<n_GPU)
	{
		if(numDevices == 1)
			fprintf(stderr, "Error: MPI rank %i: Cannot allocate %i GPUs, because only one is available\n", rank, n_GPU);
		else
			fprintf(stderr, "Error: MPI rank %i: Cannot allocate %i GPUs, because only %i are available\n", rank, n_GPU, numDevices);
		n_GPU = numDevices;
		printf("Number of GPUs set to %i\n", n_GPU);
	}

	//Converting the Nx3 matrix to 3Nx1 vector.
	if(!(xx_tmp = (REAL*)malloc(N*sizeof(REAL))))
	{
		fprintf(stderr, "MPI task %i: failed to allocate memory for xx_tmp (for CUDA force canculation).\n", rank);
		exit(-2);
	}
	if(!(xy_tmp = (REAL*)malloc(N*sizeof(REAL))))
	{
		fprintf(stderr, "MPI task %i: failed to allocate memory for xx_tmp (for CUDA force canculation).\n", rank);
		exit(-2);
	}
	if(!(xz_tmp = (REAL*)malloc(N*sizeof(REAL))))
	{
		fprintf(stderr, "MPI task %i: failed to allocate memory for xx_tmp (for CUDA force canculation).\n", rank);
		exit(-2);
	}
	for(i = 0; i < N; i++)
	{
		xx_tmp[i] = x[3*i];
		xy_tmp[i] = x[3*i+1];
		xz_tmp[i] = x[3*i+2];
	}
	//timing
	omp_start_time = omp_get_wtime();
        //timing
	omp_set_dynamic(0);             // Explicitly disable dynamic teams
	omp_set_num_threads(n_GPU);     // Use n_GPU threads
#pragma omp parallel default(shared) private(GPU_ID, F_tmp, i, j, mprocessors, cudaStatus, N_GPU, GPU_index_min, nthreads, dev_xx, dev_xy, dev_xz, dev_M, dev_F, dev_SOFT_LENGTH, dev_e)
{
		#pragma omp critical
		{
		nthreads = omp_get_num_threads();
		GPU_ID = omp_get_thread_num(); //thread ID = GPU_ID
		}
		if(GPU_ID == 0)
		{
			N_GPU = (ID_max-ID_min+1)/n_GPU+(ID_max-ID_min+1)%n_GPU;
			GPU_index_min = ID_min;
		}
		else
		{
			N_GPU = (ID_max-ID_min+1)/n_GPU;
			GPU_index_min = ID_min + (ID_max-ID_min+1)%n_GPU+N_GPU*GPU_ID;
		}
		F_tmp = (REAL*)malloc(3 * N_GPU * sizeof(REAL));
		for(i=0; i < N_GPU; i++)
		{
			for(j=0; j<3; j++)
				F_tmp[3*i + j] = 0.0f;
		}
		//Checking for the GPU
		#pragma omp critical
		hipDeviceGetAttribute(&mprocessors, hipDeviceAttributeMultiprocessorCount, GPU_ID);
		if(GPU_ID == 0)
		{
			printf("MPI task %i: GPU force calculation.\n Number of GPUs: %i\n Number of OMP threads: %i\n Number of threads per GPU: %i\n", rank, n_GPU, nthreads, 32*mprocessors*BLOCKSIZE);
		}
		#pragma omp critical
		cudaStatus = hipSetDevice(GPU_ID); //selecting GPU
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		// Allocate GPU buffers for coordinate and mass vectors
		cudaStatus = hipMalloc((void**)&dev_xx, N * sizeof(REAL));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: xx hipMalloc failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&dev_xy, N * sizeof(REAL));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: xy hipMalloc failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&dev_xz, N * sizeof(REAL));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: xz hipMalloc failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&dev_M, N * sizeof(REAL));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: M hipMalloc failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		// Allocate GPU buffers for the softening vector
		cudaStatus = hipMalloc((void**)&dev_SOFT_LENGTH, N * sizeof(REAL)); //v0.3.7.1
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: SOFT_LENGTH hipMalloc failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		// Allocate GPU buffers for force vectors
		cudaStatus = hipMalloc((void**)&dev_F, 3 * N_GPU * sizeof(REAL));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: F hipMalloc failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		// Allocate GPU buffers for e matrix
		cudaStatus = hipMalloc((void**)&dev_e, 6606 * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: e hipMalloc failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		//Converting e matrix into a vector
		for (i = 0; i < 2202; i++)
		{
			for (j = 0; j < 3; j++)
			{
				e_tmp[3 * i + j] = e[i][j];
			}
		}
		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_xx, xx_tmp, N * sizeof(REAL), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: hipMemcpy xx in failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		cudaStatus = hipMemcpy(dev_xy, xy_tmp, N * sizeof(REAL), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: hipMemcpy xy in failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		cudaStatus = hipMemcpy(dev_xz, xz_tmp, N * sizeof(REAL), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: hipMemcpy xz in failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		cudaStatus = hipMemcpy(dev_M, M, N * sizeof(REAL), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: hipMemcpy M in failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		cudaStatus = hipMemcpy(dev_SOFT_LENGTH, SOFT_LENGTH, N * sizeof(REAL), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: hipMemcpy SOFT_LENGTH in failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		cudaStatus = hipMemcpy(dev_F, F_tmp, 3 * N_GPU * sizeof(REAL), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: hipMemcpy F in failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		cudaStatus = hipMemcpy(dev_e, e_tmp, 6606 * sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: hipMemcpy e in failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		printf("MPI task %i: GPU%i: ID_min = %i\tID_max = %i\n", rank, GPU_ID, GPU_index_min, GPU_index_min+N_GPU-1);
		// Launch a kernel on the GPU with one thread for each element.
		ForceKernel_periodic << <32*mprocessors, BLOCKSIZE>> >(32*mprocessors * BLOCKSIZE, N, dev_xx, dev_xy, dev_xz, dev_F, IS_PERIODIC, dev_M, dev_SOFT_LENGTH, L, dev_e, el, GPU_index_min, GPU_index_min+N_GPU-1);

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: ForceKernel_periodic launch failed: %s\n", rank, GPU_ID, hipGetErrorString(cudaStatus));
			ForceError = true;
			goto Error;
		}
		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: hipDeviceSynchronize returned error code %d after launching ForceKernel_periodic!\n", rank, GPU_ID, cudaStatus);
			ForceError = true;
			goto Error;
		}
		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(F_tmp, dev_F, 3 * N_GPU * sizeof(REAL), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "MPI rank %i: GPU%i: hipMemcpy out failed!\n", rank, GPU_ID);
			ForceError = true;
			goto Error;
		}
		if(GPU_ID == 0)
		{
			for (i = 0; i < N_GPU; i++)
			{
				for (j = 0; j < 3; j++)
				{
					F[3*i+j] = F_tmp[(3 * i) + j];
				}
			}
		}
		else
		{
			for (i = GPU_index_min; i < GPU_index_min + N_GPU; i++)
			{
				for (j = 0; j < 3; j++)
				{
					F[3*(i-ID_min)+j] = F_tmp[3 * (i-GPU_index_min) + j];
				}
			}
		}
		free(F_tmp);
	Error:
		hipFree(dev_xx);
		hipFree(dev_xy);
		hipFree(dev_xz);
		hipFree(dev_M);
		hipFree(dev_F);
		hipFree(dev_SOFT_LENGTH);
		hipFree(dev_e);
		hipDeviceReset();
}
	free(xx_tmp);
	free(xy_tmp);
	free(xz_tmp);
	//timing
	omp_end_time = omp_get_wtime();
	//timing
	printf("Force calculation finished on MPI task %i. Force calculation wall-clock time = %fs.\n", rank, omp_end_time-omp_start_time);
	return cudaStatus;
}
#endif
